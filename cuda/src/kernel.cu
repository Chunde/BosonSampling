
#include <hip/hip_runtime.h>
#include <cstdint>

// Turn shared memory pointer into uint32_t address.
__device__ __forceinline__
uint32_t smem_u32addr(const void *smem_ptr) {
    uint32_t addr;
    asm ("{.reg .u64 u64addr;\n"
         " cvta.to.shared.u64 u64addr, %1;\n"
         " cvt.u32.u64 %0, u64addr;}\n"
         : "=r"(addr)
         : "l"(smem_ptr)
    );

    return addr;
}

// Non-coheret load from global memory to register
__device__ __forceinline__
void ldg32_nc(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void ldg32_nc_0(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @!p mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

// Set global memory from register
__device__ __forceinline__
void stg32(const float &reg, void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @p st.global.f32 [%0], %1;}\n"
        : : "l"(ptr), "f"(reg), "r"((int)guard)
    );
}

// Load from shared memory to register
__device__ __forceinline__
void lds32(float &reg,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.f32 {%0}, [%1];\n"
        : "=f"(reg)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds32(int &reg,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.s32 {%0}, [%1];\n"
        : "=r"(reg)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds64(float &reg0, float &reg1,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v2.f32 {%0, %1}, [%2];\n"
        : "=f"(reg0), "=f"(reg1)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds128(float &reg0, float &reg1,
            float &reg2, float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
        : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
        : "r"(addr)
    );
}

// Set shared memory from register
__device__ __forceinline__
void sts32(const float &reg, const uint32_t &addr) {
    asm volatile (
        "st.shared.f32 [%0], %1;\n"
        : : "r"(addr), "f"(reg)
    );
}

__device__ __forceinline__
void sts64(const float &reg0, const float &reg1,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v2.f32 [%0], {%1, %2};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1)
    );
}

__device__ __forceinline__
void sts128(const float &reg0, const float &reg1,
            const float &reg2, const float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3)
    );
}

/*
 * matrix Glc, Gcr and T: row-major
 *
 * mma block:
 * thread block tile: m128n128k8
 * warp tile: m32n64k8
 * thread tile: m8n8k8
 * thread fragment:
 *     matrixGlc: 8x1 FP32
 *     matrixGcr: 1x8 FP32
 *
 * ----------------------------------------------------------------
 * thread block tile map:
 *
 *                                64
 *                    --|---------------------|
 *           Gcr_tile  8|                     |
 *                    --|---------------------|
 *
 *  Glc_tile | 8 |      |    32    |
 *         --|---|    --|----------|----------|
 *           |   |    32|  warp_0  |  warp_1  |
 *           |   |    --|----------|----------|
 *           |   |      |  warp_2  |  warp_3  |
 *        128|   |      |----------|----------|
 *           |   |      |  warp_4  |  warp_5  |
 *           |   |      |----------|----------|
 *           |   |      |  warp_6  |  warp_7  |
 *         --|---|      |----------|----------|
 *
 * ----------------------------------------------------------------
 * warp tile map:
 *
 * 'z' thread map to avoid LDS.128 shared memory broadcast limitation.
 *
 *              |              16               ||
 *   Gcr_frag --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 *             1|///|   |   |   |   |   |   |   ||///|   |   |   |   |   |   |   |
 *            --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 * Glc_frag     | 2 |                           ||
 *    | 1 |                                     ||
 *  --|---|--   |---|---|---|---|---|---|---|---||---|---------------------------|
 *    |///|4    |t0 |t2 |t4 |t6 |t8 |t10|t12|t14||t0 |                           |
 *    |---|--   |---|---|---|---|---|---|---|---||---|                           |
 *    |   |     |t1 |t3 |t5 |t7 |t9 |t11|t13|t15||                               |
 *  16|---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t16|t18|t20|t22|t24|t26|t28|t30||                               |
 *    |---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t17|t19|t21|t23|t25|t27|t29|t31||                               |
 *  ==|===|=====|===|===|===|===|===|===|===|===||===|============================
 *    |///|     |t0 |                           ||t0 |                           |
 *    |---|     |---|                           ||---|                           |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |-------------------------------||-------------------------------|
 *
 */
// CUDA kernel
__global__ __launch_bounds__(256, 2)
void kernel(const int d,
            const int tau,
            const float *U_r,
            const float *U_i,
            const float *Glc_r,
            const float *Glc_i,
            const float *Gcr_r,
            const float *Gcr_i,
            const float *LL,
            const float *LC,
            const float *LR,
            const int *CL,
            const int *CC,
            const int *CR,
            const int *incC,
            float *T_r,
            float *T_i,
            uint32_t m,
            uint32_t n,
            uint32_t k,
            uint32_t Glc_ldg_step,    // k * sizeof(float)
            uint32_t Gcr_ldg_step) {  // n * sizeof(float) * 8
    /*
     * matrix Glc & Gcr thread block tile shared memory (double buffer)
     * matrix Glc: 132 * 8 * 4Byte/item * double buffer = 4.125KB * 2
     * matrix Gcr: 64 * 8 * 4Byte/item * double buffer = 4KB
     *
     * for double buffer faster switch, Glc_smem requires 8KB * 2 shared memory
     * and 16KB aligned, Gcr_smem should be 8KB aligned, then the double buffer
     * can be switched by only 1 xor instruction:
     *     (uint32_t &)Glc_smem ^= 0x2000;
     *     (uint32_t &)Gcr_smem ^= 0x0800;
     */

// Shared memory declaration
    __shared__ __align__(32 * 1024) char smem[42 * 1024];
    float *Glc_r_smem = reinterpret_cast<float *>(smem);
    float *Glc_i_smem = reinterpret_cast<float *>(smem + 16 * 1024);
    float *Gcr_r_smem = reinterpret_cast<float *>(smem + 32 * 1024);
    float *Gcr_i_smem = reinterpret_cast<float *>(smem + 36 * 1024);
    float *LC_smem = reinterpret_cast<float *>(smem + 40 * 1024);
    float *incC_smem = reinterpret_cast<float *>(smem + 41 * 1024);

    // Glc, Gcr, U and T register fragment
    float Glc_r_frag[2][8];
    float Glc_i_frag[2][8];
    float Gcr_r_frag[2][4];
    float Gcr_i_frag[2][4];
    float U_r_frag[2][2];
    float U_i_frag[2][2];
    float T_r_frag[8][4];
    float T_i_frag[8][4];
    #pragma unroll
    for (int i = 0; i < 2; ++i) {
        #pragma unroll
        for (int j = 0; j < 2; ++j) {
            U_r_frag[i][j] = 0;
            U_i_frag[i][j] = 0;
        }
    }
    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            T_r_frag[i][j] = 0;
            T_i_frag[i][j] = 0;
        }
    }

    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t warp_id = threadIdx.x / 32;

    // 4x8 threads each warp for FFMA
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);

    // Load left and right charge values to register
    int m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + mma_tid_y * 4;
    int n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + mma_tid_x * 2;
    int cl[2]; int cr[2];
    cl[0] = CL[m_idx];
    cl[1] = CL[m_idx + 16];
    cr[0] = CR[n_idx];
    cr[1] = CR[n_idx + 16];

    // Gamma tile ldg (load from global) pointer
    const char *Glc_r_ldg_ptr = (const char *)(
        Glc_r + (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8); // 32 x 8
    const char *Glc_i_ldg_ptr = (const char *)(
        Glc_i + (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8); // 32 x 8
    const char *Gcr_r_ldg_ptr = (const char *)(
        Gcr_r + (threadIdx.x / 32) * n + blockIdx.x * 64 + threadIdx.x % 32); // 8 x 32
    const char *Gcr_i_ldg_ptr = (const char *)(
        Gcr_i + (threadIdx.x / 32) * n + blockIdx.x * 64 + threadIdx.x % 32); // 8 x 32
    // Center Lambda and charge increment index ldg pointer
    const char *LC_ldg_ptr = (const char *)(LC + threadIdx.x);
    const char *incC_ldg_ptr = (const char *)(incC + threadIdx.x);

    // Gamma tile sts/lds (set shared memory/load shared memory) pointer
    // using uint32_t pointer for faster double buffer switch
    uint32_t Glc_r_sts_addr = smem_u32addr(
        Glc_r_smem + (threadIdx.x % 8) * 132 + (threadIdx.x / 8) * 4);
    uint32_t Glc_i_sts_addr = smem_u32addr(
        Glc_i_smem + (threadIdx.x % 8) * 132 + (threadIdx.x / 8) * 4);
    uint32_t Gcr_r_sts_addr = smem_u32addr(
        Gcr_r_smem + (threadIdx.x / 32) * 64 + (threadIdx.x % 32));
    uint32_t Gcr_i_sts_addr = smem_u32addr(
        Gcr_i_smem + (threadIdx.x / 32) * 64 + (threadIdx.x % 32));
    uint32_t LC_sts_addr = smem_u32addr(LC_smem + threadIdx.x);
    uint32_t incC_sts_addr = smem_u32addr(incC_smem + threadIdx.x);

    uint32_t Glc_r_lds_addr = smem_u32addr(
        Glc_r_smem + (warp_id / 2) * 32 + mma_tid_y * 4);
    uint32_t Glc_i_lds_addr = smem_u32addr(
        Glc_i_smem + (warp_id / 2) * 32 + mma_tid_y * 4);
    uint32_t Gcr_r_lds_addr = smem_u32addr(
        Gcr_r_smem + (warp_id % 2) * 32 + mma_tid_x * 2);
    uint32_t Gcr_i_lds_addr = smem_u32addr(
        Gcr_i_smem + (warp_id % 2) * 32 + mma_tid_x * 2);
    uint32_t LC_lds_addr = smem_u32addr(LC_smem);
    uint32_t incC_lds_addr = smem_u32addr(incC_smem);

    // ldg_guard to avoid LDG out of bound
    uint32_t Glc_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int m_idx = blockIdx.y * 128 + threadIdx.x / 8 * 4 + i;
        if (m_idx < m) {
            Glc_ldg_guard |= (1u << i);
        }
    }
    uint32_t Gcr_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 2; ++i) {
        int n_idx = blockIdx.x * 64 + threadIdx.x % 32 + i * 32;
        if (n_idx < n) {
            Gcr_ldg_guard |= (1u << i);
        }
    }

    // Register to store values loaded from global memory before putting them into shared memory
    float Glc_r_ldg_reg[4];
    float Glc_i_ldg_reg[4];
    float Gcr_r_ldg_reg[2];
    float Gcr_i_ldg_reg[2];
    float LC_ldg_reg = 0;
    float incC_ldg_reg = 0;

    // 1'st Glc & Gcr tile loaded before the k_tile loop
    uint32_t k_tiles = (k + 7) / 8 - 1;
    uint32_t first_k_tile = k - k_tiles * 8;

    // load 1'st tile to shared memory
    { 
        // Glc
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            bool guard = (Glc_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x % 8 < first_k_tile;
            ldg32_nc_0(Glc_r_ldg_reg[i],
                       Glc_r_ldg_ptr + i * Glc_ldg_step,
                       guard);
            ldg32_nc_0(Glc_i_ldg_reg[i],
                       Glc_i_ldg_ptr + i * Glc_ldg_step,
                       guard);
        }
        sts128(Glc_r_ldg_reg[0], Glc_r_ldg_reg[1], Glc_r_ldg_reg[2], Glc_r_ldg_reg[3],
               Glc_r_sts_addr);
        sts128(Glc_i_ldg_reg[0], Glc_i_ldg_reg[1], Glc_i_ldg_reg[2], Glc_i_ldg_reg[3],
               Glc_i_sts_addr);
        // Gcr
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            bool guard = (Gcr_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x / 32 < first_k_tile;
            ldg32_nc_0(Gcr_r_ldg_reg[i],
                       Gcr_r_ldg_ptr + i * 32 * sizeof(float),
                       guard);
            ldg32_nc_0(Gcr_i_ldg_reg[i],
                       Gcr_i_ldg_ptr + i * 32 * sizeof(float),
                       guard);
        }
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            sts32(Gcr_r_ldg_reg[i], Gcr_r_sts_addr + i * 32 * sizeof(float));
            sts32(Gcr_i_ldg_reg[i], Gcr_i_sts_addr + i * 32 * sizeof(float));
        }
        // LC
        bool guard = threadIdx.x < k;
        ldg32_nc_0(LC_ldg_reg, LC_ldg_ptr, guard);
        sts32(LC_ldg_reg, LC_sts_addr);
        // CC
        guard =  threadIdx.x < d;
        ldg32_nc_0(incC_ldg_reg, incC_ldg_ptr, guard);
        sts32(incC_ldg_reg, incC_sts_addr);

        __syncthreads();

        // switch double buffer
        Glc_r_sts_addr ^= 0x2000;
        Glc_i_sts_addr ^= 0x2000;
        Gcr_r_sts_addr ^= 0x0800;
        Gcr_i_sts_addr ^= 0x0800;

        // ldg pointer for next tile
        Glc_r_ldg_ptr += first_k_tile * sizeof(float);
        Glc_i_ldg_ptr += first_k_tile * sizeof(float);
        Gcr_r_ldg_ptr += n * first_k_tile * sizeof(float);
        Gcr_i_ldg_ptr += n * first_k_tile * sizeof(float);
    }

    // load 1'st fragment
    lds128(Glc_r_frag[0][0], Glc_r_frag[0][1], Glc_r_frag[0][2], Glc_r_frag[0][3],
           Glc_r_lds_addr);
    lds128(Glc_i_frag[0][0], Glc_i_frag[0][1], Glc_i_frag[0][2], Glc_i_frag[0][3],
           Glc_i_lds_addr);
    lds128(Glc_r_frag[0][4], Glc_r_frag[0][5], Glc_r_frag[0][6], Glc_r_frag[0][7],
           Glc_r_lds_addr + 16 * sizeof(float));
    lds128(Glc_i_frag[0][4], Glc_i_frag[0][5], Glc_i_frag[0][6], Glc_i_frag[0][7],
           Glc_i_lds_addr + 16 * sizeof(float));
    lds64(Gcr_r_frag[0][0], Gcr_r_frag[0][1],
          Gcr_r_lds_addr);
    lds64(Gcr_i_frag[0][0], Gcr_i_frag[0][1],
          Gcr_i_lds_addr);
    lds64(Gcr_r_frag[0][2], Gcr_r_frag[0][3],
          Gcr_r_lds_addr + 16 * sizeof(float));
    lds64(Gcr_i_frag[0][2], Gcr_i_frag[0][3],
          Gcr_i_lds_addr + 16 * sizeof(float));
    // Load the center singular value lambda center
    int c = 0;
    uint8_t c_rem = 0;
    float lc[2];
    lds32(lc[0], LC_lds_addr);
    // Load U
    #pragma unroll
    for (int i = 0; i < 2; ++i) {
        for (int j = 0; j < 2; ++j) {
            if (cl[i] >= 0 && 0 >= cr[j]) {
                U_r_frag[i][j] = U_r[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i]];
                U_i_frag[i][j] = U_i[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i]];
            }
            else { U_r_frag[i][j] = U_i_frag[i][j] = 0; }
        }
    }
    __syncthreads();
    // Find the next center charge and index that is higher than the current charge
    /* Charge array:
    *  |---|---|---|---|---|---|---|---|---|
    *  | 0 | 0 | 0 | 2 | 2 | 3 | 4 | 6 | 6 |  ...
    *  |---|---|---|---|---|---|---|---|---|
    *  incC array: If no indices corresponds to a charge value, incC = -1 at that charge value
    *  |---|---|---|---|---|---|---|
    *  | 0 |-1 | 3 | 5 | 6 |-1 | 8 | ...
    *  |---|---|---|---|---|---|---|            */
    int cc = 0;
    int old_cc = cc;
    int incCidx = 0;

    // k_tiles loop
    for (int k_tile = k_tiles; k_tile > 0; --k_tile) {
        #pragma unroll
        for (int k_frag = 0; k_frag < 8; ++k_frag) {
            if (k_tile < k_tiles || k_frag < first_k_tile){
                // Load next center charge increment index
                if (c == incCidx) {
                    old_cc = cc;
                    cc++;
                    incCidx = -1;
                    for (; cc < d && incCidx <= 0; ++cc) {
                        incCidx = incC[cc];
                        if (incCidx == 0) { old_cc = cc; }
                    }
                    cc--;
                    // Load needed U fragments
                    #pragma unroll
                    for (int i = 0; i < 2; ++i) {
                        for (int j = 0; j < 2; ++j) {
                            if (cl[i] >= old_cc && old_cc >= cr[j]) {
                                U_r_frag[i][j] = U_r[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i] - old_cc];
                                U_i_frag[i][j] = U_i[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i] - old_cc];
                            }
                            else { U_r_frag[i][j] = U_i_frag[i][j] = 0; }
                        }
                    }
                    __syncthreads();
                }
                // Increase c
                c += 1;
                c_rem += 1;
                // Load center Lambda into shared memory every 256 index iterations
                if (c_rem == 0) {
                    bool guard = threadIdx.x < (k - 256 * c/256);
                    ldg32_nc(LC_ldg_reg, LC_ldg_ptr + 256 * c/256 * sizeof(float), guard);
                    __syncthreads();
                    sts32(LC_ldg_reg, LC_sts_addr);
                    __syncthreads();
                }
            }

            // store next Glc&Gcr tile to shared memory
            if (k_frag == 7) {
                sts128(Glc_r_ldg_reg[0], Glc_r_ldg_reg[1], Glc_r_ldg_reg[2], Glc_r_ldg_reg[3],
                       Glc_r_sts_addr);
                sts128(Glc_i_ldg_reg[0], Glc_i_ldg_reg[1], Glc_i_ldg_reg[2], Glc_i_ldg_reg[3],
                       Glc_i_sts_addr);
                #pragma unroll
                for (int i = 0; i < 2; ++i) {
                    sts32(Gcr_r_ldg_reg[i], Gcr_r_sts_addr + i * 32 * sizeof(float));
                    sts32(Gcr_i_ldg_reg[i], Gcr_i_sts_addr + i * 32 * sizeof(float));
                }
                __syncthreads();
                // switch double buffer
                Glc_r_lds_addr ^= 0x2000;
                Glc_i_lds_addr ^= 0x2000;
                Gcr_r_lds_addr ^= 0x0800;
                Gcr_i_lds_addr ^= 0x0800;
                Glc_r_sts_addr ^= 0x2000;
                Glc_i_sts_addr ^= 0x2000;
                Gcr_r_sts_addr ^= 0x0800;
                Gcr_i_sts_addr ^= 0x0800;
                // ldg pointer for next tile
                Glc_r_ldg_ptr += 8 * sizeof(float);
                Glc_i_ldg_ptr += 8 * sizeof(float);
                Gcr_r_ldg_ptr += Gcr_ldg_step;
                Gcr_i_ldg_ptr += Gcr_ldg_step;
            }

            // load next Glc&Gcr fragment from shared memory to register
            lds32(lc[(k_frag + 1) % 2], LC_lds_addr + c_rem * sizeof(float));
            lds128(Glc_r_frag[(k_frag + 1) % 2][0],
                   Glc_r_frag[(k_frag + 1) % 2][1],
                   Glc_r_frag[(k_frag + 1) % 2][2],
                   Glc_r_frag[(k_frag + 1) % 2][3],
                   Glc_r_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_i_frag[(k_frag + 1) % 2][0],
                   Glc_i_frag[(k_frag + 1) % 2][1],
                   Glc_i_frag[(k_frag + 1) % 2][2],
                   Glc_i_frag[(k_frag + 1) % 2][3],
                   Glc_i_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_r_frag[(k_frag + 1) % 2][4],
                   Glc_r_frag[(k_frag + 1) % 2][5],
                   Glc_r_frag[(k_frag + 1) % 2][6],
                   Glc_r_frag[(k_frag + 1) % 2][7],
                   Glc_r_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds128(Glc_i_frag[(k_frag + 1) % 2][4],
                   Glc_i_frag[(k_frag + 1) % 2][5],
                   Glc_i_frag[(k_frag + 1) % 2][6],
                   Glc_i_frag[(k_frag + 1) % 2][7],
                   Glc_i_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds64(Gcr_r_frag[(k_frag + 1) % 2][0],
                  Gcr_r_frag[(k_frag + 1) % 2][1],
                  Gcr_r_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_i_frag[(k_frag + 1) % 2][0],
                  Gcr_i_frag[(k_frag + 1) % 2][1],
                  Gcr_i_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_r_frag[(k_frag + 1) % 2][2],
                  Gcr_r_frag[(k_frag + 1) % 2][3],
                  Gcr_r_lds_addr + ((k_frag + 1) % 8 * 64 + 16) * sizeof(float));
            lds64(Gcr_i_frag[(k_frag + 1) % 2][2],
                  Gcr_i_frag[(k_frag + 1) % 2][3],
                  Gcr_i_lds_addr + ((k_frag + 1) % 8 * 64 + 16) * sizeof(float));

            // load next Glc&Gcr tile
            if (k_frag == 0) {
                #pragma unroll
                for (int i = 0; i < 4; ++i) {
                    ldg32_nc(Glc_r_ldg_reg[i],
                             Glc_r_ldg_ptr + i * Glc_ldg_step,
                             (Glc_ldg_guard & (1u << i)) != 0);
                    ldg32_nc(Glc_i_ldg_reg[i],
                             Glc_i_ldg_ptr + i * Glc_ldg_step,
                             (Glc_ldg_guard & (1u << i)) != 0);
                }

                #pragma unroll
                for (int i = 0; i < 2; ++i) {
                    ldg32_nc(Gcr_r_ldg_reg[i],
                             Gcr_r_ldg_ptr + i * 32 * sizeof(float),
                             (Gcr_ldg_guard & (1u << i)) != 0);
                    ldg32_nc(Gcr_i_ldg_reg[i],
                             Gcr_i_ldg_ptr + i * 32 * sizeof(float),
                             (Gcr_ldg_guard & (1u << i)) != 0);
                }
            }
            
            // 
            if (k_tile < k_tiles || k_frag < first_k_tile){
                // FFMA loop
                #pragma unroll
                for (int i = 0; i < 8; ++i) {
                    #pragma unroll
                    for (int j = 0; j < 4; ++j) {
                        T_r_frag[i][j] += lc[k_frag % 2] * (U_r_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j] - Glc_i_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j]) - U_i_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j] + Glc_i_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j]));
                        T_i_frag[i][j] += lc[k_frag % 2] * (U_r_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j] + Glc_i_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j]) + U_i_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j] - Glc_i_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j]));
                    }
                }
            }
        }
    }

    // FFMA for the last tile
    #pragma unroll
    for (int k_frag = 0; k_frag < 8; ++k_frag) {
        // Load next center charge increment index
        if (c == incCidx) {
            old_cc = cc;
            cc++;
            incCidx = -1;
            for (; cc < d && incCidx <= 0; ++cc) {
                incCidx = incC[cc];
                if (incCidx == 0) { old_cc = cc; }
            }
            cc--;
            // Load needed U fragments
            #pragma unroll
            for (int i = 0; i < 2; ++i) {
                for (int j = 0; j < 2; ++j) {
                    if (cl[i] >= old_cc && old_cc >= cr[j]) {
                       U_r_frag[i][j] = U_r[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i] - old_cc];
                       U_i_frag[i][j] = U_i[(cl[i] - tau) * d * d + (tau - cr[j]) * d + cl[i] - old_cc];
                    }
                    else { U_r_frag[i][j] = 0; }
                }
            }
        }
        // Increase c
        c += 1;
        c_rem += 1;
        // Load center Lambda into shared memory every 256 index iterations
        if (c_rem == 0){
            bool guard = threadIdx.x < (k - 256 * c/256);
            ldg32_nc(LC_ldg_reg, LC_ldg_ptr + 256 * c/256 * sizeof(float), guard);
            sts32(LC_ldg_reg, LC_sts_addr);
            __syncthreads();
        }

        if (k_frag < 7) {
            // load next Glc&Gcr fragment from shared memory to register
            lds32(lc[(k_frag + 1) % 2], LC_lds_addr + c_rem * sizeof(float));
            lds128(Glc_r_frag[(k_frag + 1) % 2][0],
                   Glc_r_frag[(k_frag + 1) % 2][1],
                   Glc_r_frag[(k_frag + 1) % 2][2],
                   Glc_r_frag[(k_frag + 1) % 2][3],
                   Glc_r_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_i_frag[(k_frag + 1) % 2][0],
                   Glc_i_frag[(k_frag + 1) % 2][1],
                   Glc_i_frag[(k_frag + 1) % 2][2],
                   Glc_i_frag[(k_frag + 1) % 2][3],
                   Glc_i_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_r_frag[(k_frag + 1) % 2][4],
                   Glc_r_frag[(k_frag + 1) % 2][5],
                   Glc_r_frag[(k_frag + 1) % 2][6],
                   Glc_r_frag[(k_frag + 1) % 2][7],
                   Glc_r_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds128(Glc_i_frag[(k_frag + 1) % 2][4],
                   Glc_i_frag[(k_frag + 1) % 2][5],
                   Glc_i_frag[(k_frag + 1) % 2][6],
                   Glc_i_frag[(k_frag + 1) % 2][7],
                   Glc_i_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds64(Gcr_r_frag[(k_frag + 1) % 2][0],
                  Gcr_r_frag[(k_frag + 1) % 2][1],
                  Gcr_r_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_i_frag[(k_frag + 1) % 2][0],
                  Gcr_i_frag[(k_frag + 1) % 2][1],
                  Gcr_i_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_r_frag[(k_frag + 1) % 2][2],
                  Gcr_r_frag[(k_frag + 1) % 2][3],
                  Gcr_r_lds_addr + ((k_frag + 1) % 8 * 64 + 16) * sizeof(float));
            lds64(Gcr_i_frag[(k_frag + 1) % 2][2],
                  Gcr_i_frag[(k_frag + 1) % 2][3],
                  Gcr_i_lds_addr + ((k_frag + 1) % 8 * 64 + 16) * sizeof(float));
        }

        // FFMA loop
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                T_r_frag[i][j] += lc[k_frag % 2] * (U_r_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j] - Glc_i_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j]) - U_i_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j] + Glc_i_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j]));
                T_i_frag[i][j] += lc[k_frag % 2] * (U_r_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j] + Glc_i_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j]) + U_i_frag[i / 4][j / 2] * (Glc_r_frag[k_frag % 2][i] * Gcr_r_frag[k_frag % 2][j] - Glc_i_frag[k_frag % 2][i] * Gcr_i_frag[k_frag % 2][j]));
            }
        }
    }

    // Load lambdas to register (reusing Glc_frag Gcr_frag)
    m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + mma_tid_y * 4;
    n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + mma_tid_x * 2;
    #pragma unroll
    for (int tile_y = 0; tile_y < 2; ++tile_y){
        #pragma unroll
        for (int i = 0; i < 4; ++i){
            Glc_r_frag[0][tile_y * 4 + i] = LL[m_idx + tile_y * 16 + i];
        }
    }
    #pragma unroll
    for (int tile_x = 0; tile_x < 2; ++tile_x){
        #pragma unroll
        for (int j = 0; j < 2; ++j){
            Gcr_r_frag[0][tile_x * 2 + j] = LR[n_idx + tile_x * 16 + j];
        }
    }
    // Multiply accumulator by lambda values
    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            T_r_frag[i][j] *= Glc_r_frag[0][i] *
                            Gcr_r_frag[0][j];
            T_i_frag[i][j] *= Glc_r_frag[0][i] *
                            Gcr_r_frag[0][j];
        }
    }

    // T_tile write back, reuse Glc&Gcr tile shared memory buffer
    uint32_t T_r_sts_addr = smem_u32addr((float2 *)(smem + warp_id * 1024) +
                                       mma_tid_y * 4 * 8 + mma_tid_x);
    uint32_t T_i_sts_addr = smem_u32addr((float2 *)(smem + 8192 + warp_id * 1024) +
                                       mma_tid_y * 4 * 8 + mma_tid_x);
    const float *T_r_lds_ptr = (float *)(smem + warp_id * 1024) + lane_id;
    const float *T_i_lds_ptr = (float *)(smem + 8192 + warp_id * 1024) + lane_id;

    m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + lane_id / 16;
    n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + lane_id % 16;

    float *T_r_stg_ptr = T_r + m_idx * n + n_idx;
    float *T_i_stg_ptr = T_i + m_idx * n + n_idx;

    if (m_idx >= m) {
        return;
    } else { 
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;
        uint32_t m_guard;

        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            m_guard = m < m_idx + 16 * i ? 0 : m - (m_idx + 16 * i);
            m_guard = (m_guard + 1) / 2;

            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts64(T_r_frag[i * 4 + p][j * 2],
                        T_r_frag[i * 4 + p][j * 2 + 1],
                        T_r_sts_addr + p * 8 * sizeof(float2));
                    sts64(T_i_frag[i * 4 + p][j * 2],
                        T_i_frag[i * 4 + p][j * 2 + 1],
                        T_i_sts_addr + p * 8 * sizeof(float2));
                }

                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 8; ++p) {
                    stg32(T_r_lds_ptr[p * 32],
                        T_r_stg_ptr + (i * 16 + p * 2) * n + j * 16,
                        p < m_guard && j * 16 < n_guard);
                    stg32(T_i_lds_ptr[p * 32],
                        T_i_stg_ptr + (i * 16 + p * 2) * n + j * 16,
                        p < m_guard && j * 16 < n_guard);
                }
            }
        }
    }
}