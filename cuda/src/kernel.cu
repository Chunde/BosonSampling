
#include <hip/hip_runtime.h>
#include <cstdint>

// Turn shared memory pointer into uint32_t address.
__device__ __forceinline__
uint32_t smem_u32addr(const void *smem_ptr) {
    uint32_t addr;
    asm ("{.reg .u64 u64addr;\n"
         " cvta.to.shared.u64 u64addr, %1;\n"
         " cvt.u32.u64 %0, u64addr;}\n"
         : "=r"(addr)
         : "l"(smem_ptr)
    );

    return addr;
}

// Non-coheret load from global memory to register
__device__ __forceinline__
void ldg32_nc(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void ldg32_nc_0(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @!p mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

// Set global memory from register
__device__ __forceinline__
void stg32(const float &reg, void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @p st.global.f32 [%0], %1;}\n"
        : : "l"(ptr), "f"(reg), "r"((int)guard)
    );
}

// Load from shared memory to register
__device__ __forceinline__
void lds32(float &reg,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.f32 {%0}, [%1];\n"
        : "=f"(reg)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds32(int &reg,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.s32 {%0}, [%1];\n"
        : "=r"(reg)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds64(float &reg0, float &reg1,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v2.f32 {%0, %1}, [%2];\n"
        : "=f"(reg0), "=f"(reg1)
        : "r"(addr)
    );
}

__device__ __forceinline__
void lds128(float &reg0, float &reg1,
            float &reg2, float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
        : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
        : "r"(addr)
    );
}

// Set shared memory from register
__device__ __forceinline__
void sts32(const float &reg, const uint32_t &addr) {
    asm volatile (
        "st.shared.f32 [%0], %1;\n"
        : : "r"(addr), "f"(reg)
    );
}

__device__ __forceinline__
void sts64(const float &reg0, const float &reg1,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v2.f32 [%0], {%1, %2};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1)
    );
}

__device__ __forceinline__
void sts128(const float &reg0, const float &reg1,
            const float &reg2, const float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3)
    );
}

/*
 * matrix Glc, Gcr and T: row-major
 *
 * mma block:
 * thread block tile: m128n128k8
 * warp tile: m32n64k8
 * thread tile: m8n8k8
 * thread fragment:
 *     matrixGlc: 8x1 FP32
 *     matrixGcr: 1x8 FP32
 *
 * ----------------------------------------------------------------
 * thread block tile map:
 *
 *                                64
 *                    --|---------------------|
 *           Gcr_tile  8|                     |
 *                    --|---------------------|
 *
 *  Glc_tile | 8 |      |    32    |
 *         --|---|    --|----------|----------|
 *           |   |    32|  warp_0  |  warp_1  |
 *           |   |    --|----------|----------|
 *           |   |      |  warp_2  |  warp_3  |
 *        128|   |      |----------|----------|
 *           |   |      |  warp_4  |  warp_5  |
 *           |   |      |----------|----------|
 *           |   |      |  warp_6  |  warp_7  |
 *         --|---|      |----------|----------|
 *
 * ----------------------------------------------------------------
 * warp tile map:
 *
 * 'z' thread map to avoid LDS.128 shared memory broadcast limitation.
 *
 *              |              16               ||
 *   Gcr_frag --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 *             1|///|   |   |   |   |   |   |   ||///|   |   |   |   |   |   |   |
 *            --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 * Glc_frag     | 2 |                           ||
 *    | 1 |                                     ||
 *  --|---|--   |---|---|---|---|---|---|---|---||---|---------------------------|
 *    |///|4    |t0 |t2 |t4 |t6 |t8 |t10|t12|t14||t0 |                           |
 *    |---|--   |---|---|---|---|---|---|---|---||---|                           |
 *    |   |     |t1 |t3 |t5 |t7 |t9 |t11|t13|t15||                               |
 *  16|---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t16|t18|t20|t22|t24|t26|t28|t30||                               |
 *    |---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t17|t19|t21|t23|t25|t27|t29|t31||                               |
 *  ==|===|=====|===|===|===|===|===|===|===|===||===|============================
 *    |///|     |t0 |                           ||t0 |                           |
 *    |---|     |---|                           ||---|                           |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |-------------------------------||-------------------------------|
 *
 */
__global__ __launch_bounds__(256, 2)
void kernel(const int d,
            const int tau,
            const float *U,
            const float *Glc,
            const float *Gcr,
            const float *LL,
            const float *LC,
            const float *LR,
            const int *CL,
            const int *CC,
            const int *CR,
            const int *incC,
            float *T,
            uint32_t m,
            uint32_t n,
            uint32_t k,
            uint32_t Glc_ldg_step,    // k * sizeof(float)
            uint32_t Gcr_ldg_step) {  // n * sizeof(float) * 8
    /*
     * matrix Glc & Gcr thread block tile shared memory (double buffer)
     * matrix Glc: 132 * 8 * 4Byte/item * double buffer = 4.125KB * 2
     * matrix Gcr: 64 * 8 * 4Byte/item * double buffer = 4KB
     *
     * for double buffer faster switch, Glc_smem requires 8KB * 2 shared memory
     * and 16KB aligned, Gcr_smem should be 8KB aligned, then the double buffer
     * can be switched by only 1 xor instruction:
     *     (uint32_t &)Glc_smem ^= 0x2000;
     *     (uint32_t &)Gcr_smem ^= 0x0800;
     */

    __shared__ __align__(16 * 1024) char smem[22 * 1024];
    float *Glc_smem = reinterpret_cast<float *>(smem);
    float *Gcr_smem = reinterpret_cast<float *>(smem + 16 * 1024);
    float *LC_smem = reinterpret_cast<float *>(smem + 20 * 1024);
    float *incC_smem = reinterpret_cast<float *>(smem + 21 * 1024);

    // Glc, Gcr and T register fragment
    float Glc_frag[2][8];
    float Gcr_frag[2][4];
    float T_frag[8][4];
    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            T_frag[i][j] = 0;
        }
    }

    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t warp_id = threadIdx.x / 32;

    // 4x8 threads each warp for FFMA
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);

    // Glc_tile & Gcr_tile ldg (load from global) pointer
    const char *Glc_ldg_ptr = (const char *)(
        Glc + (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8); // 32 x 8
    const char *Gcr_ldg_ptr = (const char *)(
        Gcr + (threadIdx.x / 32) * n + blockIdx.x * 64 + threadIdx.x % 32); // 8 x 32
    const char *LC_ldg_ptr = (const char *)(LC + threadIdx.x);
    const char *incC_ldg_ptr = (const char *)(incC + threadIdx.x);

    // Glc_tile & Gcr_tile sts/lds (set shared memory/load shared memory) pointer
    // using uint32_t pointer for faster double buffer switch
    uint32_t Glc_sts_addr = smem_u32addr(
        Glc_smem + (threadIdx.x % 8) * 132 + (threadIdx.x / 8) * 4);
    uint32_t Gcr_sts_addr = smem_u32addr(
        Gcr_smem + (threadIdx.x / 32) * 64 + (threadIdx.x % 32));
    uint32_t LC_sts_addr = smem_u32addr(LC_smem + threadIdx.x);
    uint32_t incC_sts_addr = smem_u32addr(incC_smem + threadIdx.x);

    uint32_t Glc_lds_addr = smem_u32addr(
        Glc_smem + (warp_id / 2) * 32 + mma_tid_y * 4);
    uint32_t Gcr_lds_addr = smem_u32addr(
        Gcr_smem + (warp_id % 2) * 32 + mma_tid_x * 2);
    uint32_t LC_lds_addr = smem_u32addr(LC_smem);
    uint32_t incC_lds_addr = smem_u32addr(incC_smem);

    // ldg_guard to avoid LDG out of bound
    uint32_t Glc_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int m_idx = blockIdx.y * 128 + threadIdx.x / 8 * 4 + i;
        if (m_idx < m) {
            Glc_ldg_guard |= (1u << i);
        }
    }
    uint32_t Gcr_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 2; ++i) {
        int n_idx = blockIdx.x * 64 + threadIdx.x % 32 + i * 32;
        if (n_idx < n) {
            Gcr_ldg_guard |= (1u << i);
        }
    }

    // Register to store values loaded from global memory before putting them into shared memory
    float Glc_ldg_reg[4];
    float Gcr_ldg_reg[2];
    float LC_ldg_reg = 0;
    float incC_ldg_reg = 0;

    // Load left and right charge values to register
    int m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + mma_tid_y * 4;
    int n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + mma_tid_x * 2;
    int cl; int cr;

    // 1'st Glc & Gcr tile loaded before the k_tile loop
    uint32_t k_tiles = (k + 7) / 8 - 1;
    uint32_t first_k_tile = k - k_tiles * 8;

    // load 1'st tile to shared memory
    { 
        // Glc
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            bool guard = (Glc_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x % 8 < first_k_tile;
            ldg32_nc_0(Glc_ldg_reg[i],
                       Glc_ldg_ptr + i * Glc_ldg_step,
                       guard);
        }
        sts128(Glc_ldg_reg[0], Glc_ldg_reg[1], Glc_ldg_reg[2], Glc_ldg_reg[3],
               Glc_sts_addr);
        // Gcr
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            bool guard = (Gcr_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x / 32 < first_k_tile;
            ldg32_nc_0(Gcr_ldg_reg[i],
                       Gcr_ldg_ptr + i * 32 * sizeof(float),
                       guard);
        }
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            sts32(Gcr_ldg_reg[i], Gcr_sts_addr + i * 32 * sizeof(float));
        }
        // LC
        bool guard = threadIdx.x < k;
        ldg32_nc_0(LC_ldg_reg, LC_ldg_ptr, guard);
        sts32(LC_ldg_reg, LC_sts_addr);
        // CC
        guard =  threadIdx.x < d;
        ldg32_nc_0(incC_ldg_reg, incC_ldg_ptr, guard);
        sts32(incC_ldg_reg, incC_sts_addr);

        __syncthreads();

        // switch double buffer
        Glc_sts_addr ^= 0x2000;
        Gcr_sts_addr ^= 0x0800;

        // ldg pointer for next tile
        Glc_ldg_ptr += first_k_tile * sizeof(float);
        Gcr_ldg_ptr += n * first_k_tile * sizeof(float);
    }

    // load 1'st fragment
    lds128(Glc_frag[0][0], Glc_frag[0][1], Glc_frag[0][2], Glc_frag[0][3],
           Glc_lds_addr);
    lds128(Glc_frag[0][4], Glc_frag[0][5], Glc_frag[0][6], Glc_frag[0][7],
           Glc_lds_addr + 16 * sizeof(float));
    lds64(Gcr_frag[0][0], Gcr_frag[0][1],
          Gcr_lds_addr);
    lds64(Gcr_frag[0][2], Gcr_frag[0][3],
          Gcr_lds_addr + 16 * sizeof(float));
    // Load the center singular value lambda center
    int c = 0;
    uint8_t c_rem = 0;
    float lc[2];
    lds32(lc[0], LC_lds_addr);
    // Find the next center charge and index that is higher than the current charge
    /* Charge array:
    *  |---|---|---|---|---|---|---|---|---|
    *  | 0 | 0 | 0 | 2 | 2 | 3 | 4 | 6 | 6 |  ...
    *  |---|---|---|---|---|---|---|---|---|
    *  incC array
    *  |---|---|---|---|---|---|---|
    *  | 0 | 0 | 3 | 5 | 6 | 0 | 8 | ...
    *  |---|---|---|---|---|---|---|            */
    int incCidx = 0;
    int cc = 0;
    // Load the first value of U needed for the center charge cc
    float u[2];
    const char *U_ldg_ptr = (const char *)(U + (cl-tau)*d*d + (tau-cr)*d + (cl-cc));
    ldg32_nc_0(u[0], U_ldg_ptr, true);

    // k_tiles loop
    for (int k_tile = k_tiles; k_tile > 0; --k_tile) {
        #pragma unroll
        for (int k_frag = 0; k_frag < 8; ++k_frag) {
            if (k_tile < k_tiles || k_frag < first_k_tile){
                c += 1;
                c_rem += 1;
                if (c_rem == 0) {
                    bool guard = threadIdx.x < (k - 256 * c/256);
                    ldg32_nc(LC_ldg_reg, LC_ldg_ptr + 256 * c/256 * sizeof(float), guard);
                    __syncthreads();
                    sts32(LC_ldg_reg, LC_sts_addr);
                    __syncthreads();
                }
                if (c == incCidx) {
                    for (; cc < d && incCidx == 0; ++cc) {
                        lds32(incCidx, incC_lds_addr + cc * sizeof(int));
                        __syncthreads();
                    }
                }
            }

            // store next Glc&Gcr tile to shared memory
            if (k_frag == 7) {
                sts128(Glc_ldg_reg[0], Glc_ldg_reg[1], Glc_ldg_reg[2], Glc_ldg_reg[3],
                       Glc_sts_addr);
                #pragma unroll
                for (int i = 0; i < 2; ++i) {
                    sts32(Gcr_ldg_reg[i], Gcr_sts_addr + i * 32 * sizeof(float));
                }
                __syncthreads();
                // switch double buffer
                Glc_lds_addr ^= 0x2000;
                Gcr_lds_addr ^= 0x0800;
                Glc_sts_addr ^= 0x2000;
                Gcr_sts_addr ^= 0x0800;
                // ldg pointer for next tile
                Glc_ldg_ptr += 8 * sizeof(float);
                Gcr_ldg_ptr += Gcr_ldg_step;
            }

            // load next Glc&Gcr fragment from shared memory to register
            lds32(lc[(k_frag + 1) % 2], LC_lds_addr + c_rem * sizeof(float));
            lds128(Glc_frag[(k_frag + 1) % 2][0],
                   Glc_frag[(k_frag + 1) % 2][1],
                   Glc_frag[(k_frag + 1) % 2][2],
                   Glc_frag[(k_frag + 1) % 2][3],
                   Glc_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_frag[(k_frag + 1) % 2][4],
                   Glc_frag[(k_frag + 1) % 2][5],
                   Glc_frag[(k_frag + 1) % 2][6],
                   Glc_frag[(k_frag + 1) % 2][7],
                   Glc_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds64(Gcr_frag[(k_frag + 1) % 2][0],
                  Gcr_frag[(k_frag + 1) % 2][1],
                  Gcr_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_frag[(k_frag + 1) % 2][2],
                  Gcr_frag[(k_frag + 1) % 2][3],
                  Gcr_lds_addr + ((k_frag + 1) % 8 * 64 + 16) * sizeof(float));

            // load next Glc&Gcr tile
            if (k_frag == 0) {
                #pragma unroll
                for (int i = 0; i < 4; ++i) {
                    ldg32_nc(Glc_ldg_reg[i],
                             Glc_ldg_ptr + i * Glc_ldg_step,
                             (Glc_ldg_guard & (1u << i)) != 0);
                }

                #pragma unroll
                for (int i = 0; i < 2; ++i) {
                    ldg32_nc(Gcr_ldg_reg[i],
                             Gcr_ldg_ptr + i * 32 * sizeof(float),
                             (Gcr_ldg_guard & (1u << i)) != 0);
                }
            }
            
            // 
            if (k_tile < k_tiles || k_frag < first_k_tile){
                // FFMA loop
                #pragma unroll
                for (int i = 0; i < 8; ++i) {
                    #pragma unroll
                    for (int j = 0; j < 4; ++j) {
                        T_frag[i][j] += Glc_frag[k_frag % 2][i] *
                                        Gcr_frag[k_frag % 2][j] *
                                        lc[k_frag % 2];
                    }
                }
            }
        }
    }

    // FFMA for the last tile
    #pragma unroll
    for (int k_frag = 0; k_frag < 8; ++k_frag) {

        c += 1;
        if (c % 256 == 0){
            bool guard = threadIdx.x < (k - 256 * c/256);
            ldg32_nc(LC_ldg_reg, LC_ldg_ptr + 256 * c/256 * sizeof(float), guard);
            sts32(LC_ldg_reg, LC_sts_addr);
            __syncthreads();
        }

        if (k_frag < 7) {
            // load next Glc&Gcr fragment from shared memory to register
            lds128(Glc_frag[(k_frag + 1) % 2][0],
                   Glc_frag[(k_frag + 1) % 2][1],
                   Glc_frag[(k_frag + 1) % 2][2],
                   Glc_frag[(k_frag + 1) % 2][3],
                   Glc_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(Glc_frag[(k_frag + 1) % 2][4],
                   Glc_frag[(k_frag + 1) % 2][5],
                   Glc_frag[(k_frag + 1) % 2][6],
                   Glc_frag[(k_frag + 1) % 2][7],
                   Glc_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds64(Gcr_frag[(k_frag + 1) % 2][0],
                  Gcr_frag[(k_frag + 1) % 2][1],
                  Gcr_lds_addr + (k_frag + 1) % 8 * 64 * sizeof(float));
            lds64(Gcr_frag[(k_frag + 1) % 2][2],
                  Gcr_frag[(k_frag + 1) % 2][3],
                  Gcr_lds_addr + ((k_frag + 1) % 8 * 64 + 16)* sizeof(float));
            lds32(lc[(k_frag + 1) % 2], LC_lds_addr + (c % 256) * sizeof(float));
        }

        // FFMA loop
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                T_frag[i][j] += Glc_frag[k_frag % 2][i] *
                                Gcr_frag[k_frag % 2][j] *
                                lc[k_frag % 2];
            }
        }
    }

    // Load lambdas to register (reusing Glc_frag Gcr_frag)
    m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + mma_tid_y * 4;
    n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + mma_tid_x * 2;
    #pragma unroll
    for (int tile_y = 0; tile_y < 2; ++tile_y){
        #pragma unroll
        for (int i = 0; i < 4; ++i){
            Glc_frag[0][tile_y * 4 + i] = LL[m_idx + tile_y * 16 + i];
        }
    }
    #pragma unroll
    for (int tile_x = 0; tile_x < 2; ++tile_x){
        #pragma unroll
        for (int j = 0; j < 2; ++j){
            Gcr_frag[0][tile_x * 2 + j] = LR[n_idx + tile_x * 16 + j];
        }
    }
    // Multiply accumulator by lambda values
    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            T_frag[i][j] *= Glc_frag[0][i] *
                            Gcr_frag[0][j];
        }
    }

    // T_tile write back, reuse Glc&Gcr tile shared memory buffer
    uint32_t T_sts_addr = smem_u32addr((float2 *)(smem + warp_id * 1024) +
                                       mma_tid_y * 4 * 8 + mma_tid_x);
    const float *T_lds_ptr = (float *)(smem + warp_id * 1024) + lane_id;

    m_idx = blockIdx.y * 128 + warp_id / 2 * 32 + lane_id / 16;
    n_idx = blockIdx.x * 64 + warp_id % 2 * 32 + lane_id % 16;

    float *T_stg_ptr = T + m_idx * n + n_idx;

    if (m_idx >= m) {
        return;
    } else { 
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;
        uint32_t m_guard;

        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            m_guard = m < m_idx + 16 * i ? 0 : m - (m_idx + 16 * i);
            m_guard = (m_guard + 1) / 2;

            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts64(T_frag[i * 4 + p][j * 2],
                        T_frag[i * 4 + p][j * 2 + 1],
                        T_sts_addr + p * 8 * sizeof(float2));
                }

                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 8; ++p) {
                    stg32(T_lds_ptr[p * 32],
                        T_stg_ptr + (i * 16 + p * 2) * n + j * 16,
                        p < m_guard && j * 16 < n_guard);
                }
            }
        }
    }
}