// How to compile: nvcc -o test test.cu -arch sm_50 -I ~/YHs_Sample/cnpy -I ./ -L/usr/local/lib -lcnpy -lz

#include"cnpy.h"

#include <TwoDInit.h>
#include <sort.h>

int main() {
    unsigned int m = 17;
    unsigned int n = 21;
    unsigned int d = 3;

    int *h_CL, *h_CR;
    hipHostMalloc(&h_CL, m * sizeof(int));
    hipHostMalloc(&h_CR, n * sizeof(int));

    random_init(h_CL, m, 0, d-1);
    random_init(h_CR, n, 0, d-1);

    int *d_CL, *d_CR;
    hipMalloc(&d_CL, m * sizeof(int));
    hipMalloc(&d_CR, n * sizeof(int));

    hipMemcpy(d_CL, h_CL, m * sizeof(int), hipMemcpyDefault);
    hipMemcpy(d_CR, h_CR, n * sizeof(int), hipMemcpyDefault);

    SortedInfo sortedL = sort(d_CL, m);
    SortedInfo sortedR = sort(d_CR, m);
    int* d_incL = sortedL.inc;
    int* d_incR = sortedR.inc;
    int *incL, *incR;
    hipHostMalloc(&incL, d * sizeof(int));
    hipHostMalloc(&incR, d * sizeof(int));
    hipMemcpy(incL, d_incL, d * sizeof(int), hipMemcpyDefault);
    hipMemcpy(incR, d_incR, d * sizeof(int), hipMemcpyDefault);

    NewData new_data = random_init_2D(m, n, d, incL, incR);
    float* data = new_data.data;
    unsigned int mNew =  new_data.m;
    unsigned int nNew = new_data.n;

    hipFree(d_CL);
    hipFree(d_CR);
    hipFree(d_incR);
    hipFree(d_incL);

    //save results to file
    cnpy::npy_save("data.npy", &data[0], {mNew, nNew}, "w");

    hipHostFree(h_CL);
    hipHostFree(h_CR);
    hipHostFree(incR);
    hipHostFree(incL);
}