// How to compile: nvcc -o test test.cu -arch sm_50 -I ~/YHs_Sample/cnpy -I ./ -L/usr/local/lib -lcnpy -lz
#include <../include/DataInit.cuh>
#include <../include/sort.cuh>
#include <../include/save.cuh>

int main() {
    unsigned int m = 100;
    unsigned int n = 21;
    unsigned int d = 10;

    int *h_CL, *h_CR;
    hipHostMalloc(&h_CL, m * sizeof(int));
    hipHostMalloc(&h_CR, n * sizeof(int));

    random_init(h_CL, m, 2, d-1);
    random_init(h_CR, n, 0, d-1);

    int *d_CL, *d_CR;
    hipMalloc(&d_CL, m * sizeof(int));
    hipMalloc(&d_CR, n * sizeof(int));

    hipMemcpy(d_CL, h_CL, m * sizeof(int), hipMemcpyDefault);
    hipMemcpy(d_CR, h_CR, n * sizeof(int), hipMemcpyDefault);

    SortedInfo sortedL = sort(d_CL, m);
    SortedInfo sortedR = sort(d_CR, m);
    int* d_incL = sortedL.inc;
    int* d_incR = sortedR.inc;
    int *incL, *incR;
    hipHostMalloc(&incL, d * sizeof(int));
    hipHostMalloc(&incR, d * sizeof(int));
    hipMemcpy(incL, d_incL, d * sizeof(int), hipMemcpyDefault);
    hipMemcpy(incR, d_incR, d * sizeof(int), hipMemcpyDefault);

    NewData LL_data = left_align_init_1d(m, d, incL);
    float* data = LL_data.data;
    unsigned int mNew = LL_data.m;

    // NewData new_data = random_init_2D(m, n, d, incL, incR);
    // float* data = new_data.data;
    // unsigned int mNew = new_data.m;
    // unsigned int nNew = new_data.n;

    hipFree(d_CL);
    hipFree(d_CR);
    hipFree(d_incR);
    hipFree(d_incL);

    //save results to file
    save((std::string)"../out/data.npy", data, mNew);
    save((std::string)"../out/CL.npy", h_CL, m);
    save((std::string)"../out/incL.npy", incL, d);


    hipHostFree(h_CL);
    hipHostFree(h_CR);
    hipHostFree(incR);
    hipHostFree(incL);
}