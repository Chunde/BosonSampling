// How to compile: nvcc -o main main.cu ./src/kernel.cu ./src/DataInit.cu ./src/check.cu ./src/sort.cu -arch sm_50 -I ~/YHs_Sample/cnpy -I ./include -L/usr/local/lib -lcnpy
#include <assert.h>

#include <cnpy.h>

#include <kernel.cuh>
#include <check.cuh>
#include <sort.cuh>
#include <DataInit.cuh>

int main() {
    unsigned int m;
    unsigned int n;
    unsigned int k;
    unsigned int d = 20;
    unsigned int tau = 10;
    unsigned int n_iter = 1;
    bool chk = true;

    for (int i = 0; i < 100 && chk; i += 100){
        m = 1000 + i;
        n = 1000 + i;
        k = 1000 + i;

        //////////////////////
        // Data preparation //
        //////////////////////
        float *h_U, *h_T;
        int *h_CL, *h_CC, *h_CR;
        hipHostMalloc(&h_U, d*d*d * sizeof(float));
        hipHostMalloc(&h_CL, m * sizeof(int));
        hipHostMalloc(&h_CC, k * sizeof(int));
        hipHostMalloc(&h_CR, n * sizeof(int));
        hipHostMalloc(&h_T, m * n * sizeof(float));
        random_init(h_U, d*d*d);
        random_init(h_CL, m, 0, d);
        random_init(h_CC, k, 0, d);
        random_init(h_CR, n, 0, d);

        float *d_U, *d_T;
        int *d_CL, *d_CC, *d_CR;
        hipMalloc(&d_U, d*d*d * sizeof(float));
        hipMalloc(&d_CL, m * sizeof(int));
        hipMalloc(&d_CC, k * sizeof(int));
        hipMalloc(&d_CR, n * sizeof(int));
        hipMalloc(&d_T, m * n * sizeof(float));

        hipMemcpy(d_U, h_U, d*d*d * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_CL, h_CL, m * sizeof(int), hipMemcpyDefault);
        hipMemcpy(d_CC, h_CC, k * sizeof(int), hipMemcpyDefault);
        hipMemcpy(d_CR, h_CR, n * sizeof(int), hipMemcpyDefault);
        /////////////////////////////
        // End of data preparation //
        /////////////////////////////

        //////////////////////////////////////////////////////////////////////////////////////////
        // Index reorganization (sorting charges and aligning charge changes to multiples of 8) //
        //////////////////////////////////////////////////////////////////////////////////////////
        // Left side
        // Sorting
        SortedInfo sortedL = sort(d_CL, m);
        int* d_incL = sortedL.inc;
        int* d_idL = sortedL.id;
        int* h_idL, *h_incL;
        hipHostMalloc(&h_idL, m * sizeof(int));
        hipHostMalloc(&h_incL, m * sizeof(int));
        hipMemcpy(h_idL, d_idL, m * sizeof(int), hipMemcpyDefault);
        hipMemcpy(h_incL, d_incL, m * sizeof(int), hipMemcpyDefault);
        // Reindexing
        RemapInfo remapL = index_remapping(m, d, h_idL, h_incL);
        unsigned int sizeNewL = remapL.size;
        int* h_indexNewL = remapL.index;
        int* h_incNewL = remapL.inc;
        int* h_cNewL = remapL.c;
        // Transfering to device
        int *d_incNewL, *d_cNewL;
        hipMalloc(&d_incNewL, d * sizeof(int));
        hipMalloc(&d_cNewL, sizeNewL * sizeof(int));
        hipMemcpy(d_incNewL, h_incNewL, d * sizeof(int), hipMemcpyDefault);
        hipMemcpy(d_cNewL, h_cNewL, sizeNewL * sizeof(int), hipMemcpyDefault);

        // Center
        // Sorting
        SortedInfo sortedC = sort(d_CC, n);
        int* d_incC = sortedC.inc;
        int* d_idC = sortedC.id;
        int *h_idC, *h_incC;
        hipHostMalloc(&h_idC, k * sizeof(int));
        hipHostMalloc(&h_incC, k * sizeof(int));
        hipMemcpy(h_idC, d_idC, k * sizeof(int), hipMemcpyDefault);
        hipMemcpy(h_incC, d_incC, k * sizeof(int), hipMemcpyDefault);
        // Reindexing
        RemapInfo remapC = index_remapping(k, d, h_idC, h_incC);
        unsigned int sizeNewC = remapC.size;
        int* h_indexNewC = remapC.index;
        int* h_incNewC = remapC.inc;
        int* h_cNewC = remapC.c;
        // Transfering to device
        int *d_incNewC, *d_cNewC;
        hipMalloc(&d_incNewC, d * sizeof(int));
        hipMalloc(&d_cNewC, sizeNewC * sizeof(int));
        hipMemcpy(d_incNewC, h_incNewC, d * sizeof(int), hipMemcpyDefault);
        hipMemcpy(d_cNewC, h_cNewC, sizeNewC * sizeof(int), hipMemcpyDefault);

        // Right side
        // Sorting
        SortedInfo sortedR = sort(d_CR, n);
        int* d_incR = sortedR.inc;
        int* d_idR = sortedR.id;
        int *h_idR, *h_incR;
        hipHostMalloc(&h_idR, n * sizeof(int));
        hipHostMalloc(&h_incR, n * sizeof(int));
        hipMemcpy(h_idR, d_idR, n * sizeof(int), hipMemcpyDefault);
        hipMemcpy(h_incR, d_incR, n * sizeof(int), hipMemcpyDefault);
        // Reindexing
        RemapInfo remapR = index_remapping(m, d, h_idR, h_incR);
        unsigned int sizeNewR = remapR.size;
        int* h_indexNewR = remapR.index;
        int* h_incNewR = remapR.inc;
        int* h_cNewR = remapR.c;
        // Transfering to device
        int *d_incNewR, *d_cNewR;
        hipMalloc(&d_incNewR, d * sizeof(int));
        hipMalloc(&d_cNewR, sizeNewR * sizeof(int));
        hipMemcpy(d_incNewR, h_incNewR, d * sizeof(int), hipMemcpyDefault);
        hipMemcpy(d_cNewR, h_cNewR, sizeNewR * sizeof(int), hipMemcpyDefault);
        ///////////////////////
        // End of reindexing //
        ///////////////////////

        ///////////////////////////////////////////////////////////////////////
        // Initializing Lambdas and Gammas according to new charge alignment //
        ///////////////////////////////////////////////////////////////////////
        // Obtaining lambdas and gammas on host
        // Lambda center doesn't need alignment
        float* h_LC;
        hipHostMalloc(&h_LC, k * sizeof(float));
        random_init(h_LC, k);
        // Lambda left, right and gammas need alignment
        NewData LL_data = left_align_init_1d(m, d, h_incL);
        NewData LR_data = right_align_init_1d(n, d, h_incR);
        NewData Glc_data = left_align_init(m, k, d, h_incL);
        NewData Gcr_data = right_align_init(k, n, d, h_incR);
        float* h_LL = LL_data.data;
        float* h_LR = LR_data.data;
        float* h_Glc = Glc_data.data;
        float* h_Gcr = Gcr_data.data;
        //assert (sizeNewL == Glc_data.m);
        //assert (sizeNewR == Gcr_data.n);
        // Moving lambdas and gammas to device
        float *d_LL, *d_LC, *d_LR, *d_Glc, *d_Gcr;
        hipMalloc(&d_LL, sizeNewL * sizeof(float));
        hipMalloc(&d_LC, k * sizeof(float));
        hipMalloc(&d_LR, sizeNewR * sizeof(float));
        hipMalloc(&d_Glc, sizeNewL * k * sizeof(float));
        hipMalloc(&d_Gcr, k * sizeNewR * sizeof(float));
        hipMemcpy(d_LL, h_LL, sizeNewL * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_LC, h_LC, k * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_LR, h_LR, sizeNewR * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_Glc, h_Glc, sizeNewL * k * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_Gcr, h_Gcr, k * sizeNewR * sizeof(float), hipMemcpyDefault);
        //cnpy::npy_save("Glc.npy", &h_Glc[0], {sizeNewL, k}, "w");
        //cnpy::npy_save("Gcr.npy", &h_Gcr[0], {k, sizeNewR}, "w");
        ////////////////////////////////////////
        // End of Lambda Gamma initialization //
        ////////////////////////////////////////

        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        dim3 grid((sizeNewR + 63) / 64, (sizeNewL + 127) / 128);

        // warmup
        kernel<<<grid, 256>>>(
            d, tau, d_U, d_Glc, d_Gcr, d_LL, d_LC, d_LR, d_cNewL, d_cNewC, d_cNewR, d_incC, d_T, sizeNewL, sizeNewR, k, k * sizeof(float), sizeNewR * sizeof(float) * 8);

        hipEventRecord(start);
        for (int i = 0; i < n_iter; ++i) {
            kernel<<<grid, 256>>>(
                d, tau, d_U, d_Glc, d_Gcr, d_LL, d_LC, d_LR, d_cNewL, d_cNewC, d_cNewR, d_incC, d_T, sizeNewL, sizeNewR, k, k * sizeof(float), sizeNewR * sizeof(float) * 8);
        }
        hipEventRecord(end);
        hipEventSynchronize(end);

        float ms;
        hipEventElapsedTime(&ms, start, end);
        hipEventDestroy(start);
        hipEventDestroy(end);

        long workload = n_iter * long(m) * n * k * 2;
        double gflops = (double(workload) / 1e9) / (double(ms) / 1e3);
        printf("Performance: %fGFLOPS; total time %fms\n", gflops, ms/n_iter);

        hipMemcpy(h_T, d_T, m * n * sizeof(float), hipMemcpyDefault);

        hipFree(d_U);
        hipFree(d_Glc);
        hipFree(d_Gcr);
        hipFree(d_LL);
        hipFree(d_LC);
        hipFree(d_LR);
        hipFree(d_T);

        chk = check(h_U, h_Glc, h_Gcr, h_LL, h_LC, h_LR, h_T, m, n, k); 

        //save results to file
        cnpy::npy_save("U.npy", &h_U[0], {d, d, d }, "w");
        cnpy::npy_save("A.npy", &h_Glc[0], {m, k}, "w");
        cnpy::npy_save("B.npy", &h_Gcr[0], {k, n}, "w");
        cnpy::npy_save("LL.npy", &h_LL[0], {m}, "w");
        cnpy::npy_save("LC.npy", &h_LC[0], {k}, "w");
        cnpy::npy_save("LR.npy", &h_LR[0], {n}, "w");
        cnpy::npy_save("C.npy", &h_T[0], {m, n}, "w");

        hipHostFree(h_U);
        hipHostFree(h_Glc);
        hipHostFree(h_Gcr);
        hipHostFree(h_LL);
        hipHostFree(h_LC);
        hipHostFree(h_LR);
        hipHostFree(h_T);
    }
}