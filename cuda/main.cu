#include "hip/hip_runtime.h"
// How to compile: nvcc -o test test.cu -arch sm_50 -I ~/YHs_Sample/cnpy -I ./ -L/usr/local/lib -lcnpy -lz
#include <assert.h>
#include <cstdio>

#include <kernel.cuh> // Kernel
#include <sort.cuh> // Sorting and index arrangement functions
#include <DataInit.cuh> // Initialization of data arrays
#include <check.cuh> // Checking the answer
#include <save.cuh> // Saving data as numpy arrays

int main() {
    unsigned int m;
    unsigned int n;
    unsigned int k;
    unsigned int d = 5;
    unsigned int tau = d/2;
    assert (d >= tau);
    unsigned int n_iter = 1;
    bool chk = true;

    for (int i = 0; i < 300 && chk; i += 3) {
        m = 5000 + i;

        for (int j = 0; j < 300; j += 3) {
            n = 5000 + j;
            d = min(m, n)/100 + 1;
            tau = d/2;
            printf("\nm: %i, n: %i, d: %i, tau: %i\n", m, n, d, tau);

            for (int l = 0; l < 3; l += 3) {
                k = 5000 + l;

                //////////////////////
                // Data preparation //
                //////////////////////
                // Initialize U
                float *h_U_r, *h_U_i;
                hipHostMalloc(&h_U_r, d*d*d * sizeof(float));
                hipHostMalloc(&h_U_i, d*d*d * sizeof(float));
                random_init(h_U_r, d*d*d);
                random_init(h_U_i, d*d*d);
                // Fill charges with random integers. A d dimensional Hilbert space has from 0 to d-1 charges possible.
                int *h_CL, *h_CC, *h_CR;
                hipHostMalloc(&h_CL, m * sizeof(int));
                hipHostMalloc(&h_CC, k * sizeof(int));
                hipHostMalloc(&h_CR, n * sizeof(int));
                random_init(h_CL, m, tau, d-1);
                random_init(h_CC, k, 0, d-1);
                random_init(h_CR, n, 0, tau);
                // Transfer data from cpu to gpu
                float *d_U_r, *d_U_i;
                hipMalloc(&d_U_r, d*d*d * sizeof(float));
                hipMalloc(&d_U_i, d*d*d * sizeof(float));
                hipMemcpy(d_U_r, h_U_r, d*d*d * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_U_i, h_U_i, d*d*d * sizeof(float), hipMemcpyDefault);
                int *d_CL, *d_CC, *d_CR;
                hipMalloc(&d_CL, m * sizeof(int));
                hipMalloc(&d_CC, k * sizeof(int));
                hipMalloc(&d_CR, n * sizeof(int));
                hipMemcpy(d_CL, h_CL, m * sizeof(int), hipMemcpyDefault);
                hipMemcpy(d_CC, h_CC, k * sizeof(int), hipMemcpyDefault);
                hipMemcpy(d_CR, h_CR, n * sizeof(int), hipMemcpyDefault);
                /////////////////////////////
                // End of data preparation //
                /////////////////////////////

                //////////////////////////////////////////////////////////////////////////////////////////
                // Index reorganization (sorting charges and aligning charge changes to multiples of 8) //
                //////////////////////////////////////////////////////////////////////////////////////////
                // Left side
                // Sorting
                SortedInfo sortedL = sort(d, m, d_CL);
                int* d_incL = sortedL.inc;
                int* d_idL = sortedL.id;
                int *h_idL, *h_incL;
                hipHostMalloc(&h_idL, m * sizeof(int));
                hipHostMalloc(&h_incL, m * sizeof(int));
                hipMemcpy(h_idL, d_idL, m * sizeof(int), hipMemcpyDefault);
                hipMemcpy(h_incL, d_incL, d * sizeof(int), hipMemcpyDefault);
                // Reindexing
                RemapInfo remapL = index_remapping(m, d, h_idL, h_incL);
                unsigned int sizeNewL = remapL.size;
                int* h_indexNewL = remapL.index;
                int* h_incNewL = remapL.inc;
                int* h_cNewL = remapL.c;
                // Transfering to device
                int *d_incNewL, *d_cNewL;
                hipMalloc(&d_incNewL, d * sizeof(int));
                hipMalloc(&d_cNewL, sizeNewL * sizeof(int));
                hipMemcpy(d_incNewL, h_incNewL, d * sizeof(int), hipMemcpyDefault);
                hipMemcpy(d_cNewL, h_cNewL, sizeNewL * sizeof(int), hipMemcpyDefault);

                // Center (doesn't need to be aligned)
                // Sorting
                SortedInfo sortedC = sort(d, k, d_CC);
                hipMemcpy(h_CC, d_CC, k * sizeof(int), hipMemcpyDefault);
                int* d_incC = sortedC.inc;
                int* d_idC = sortedC.id;
                int* h_incC;
                hipHostMalloc(&h_incC, d * sizeof(int));
                hipMemcpy(h_incC, d_incC, d * sizeof(int), hipMemcpyDefault);

                // Right side
                // Sorting
                SortedInfo sortedR = sort(d, n, d_CR);
                int* d_incR = sortedR.inc;
                int* d_idR = sortedR.id;
                int *h_idR, *h_incR;
                hipHostMalloc(&h_idR, n * sizeof(int));
                hipHostMalloc(&h_incR, n * sizeof(int));
                hipMemcpy(h_idR, d_idR, n * sizeof(int), hipMemcpyDefault);
                hipMemcpy(h_incR, d_incR, d * sizeof(int), hipMemcpyDefault);
                // Reindexing
                RemapInfo remapR = index_remapping(n, d, h_idR, h_incR);
                unsigned int sizeNewR = remapR.size;
                int* h_indexNewR = remapR.index;
                int* h_incNewR = remapR.inc;
                int* h_cNewR = remapR.c;
                // Transfering to device
                int *d_incNewR, *d_cNewR;
                hipMalloc(&d_incNewR, d * sizeof(int));
                hipMalloc(&d_cNewR, sizeNewR * sizeof(int));
                hipMemcpy(d_incNewR, h_incNewR, d * sizeof(int), hipMemcpyDefault);
                hipMemcpy(d_cNewR, h_cNewR, sizeNewR * sizeof(int), hipMemcpyDefault);
                ///////////////////////
                // End of reindexing //
                ///////////////////////

                ///////////////////////////////////////////////////////////////////////
                // Initializing Lambdas and Gammas according to new charge alignment //
                ///////////////////////////////////////////////////////////////////////
                // Obtaining lambdas and gammas on host
                // Lambda center doesn't need alignment
                float *h_LC;
                hipHostMalloc(&h_LC, k * sizeof(float));
                random_init(h_LC, k);
                // Lambda left, right and gammas need alignment
                NewData LL_data = left_align_init_1d(m, d, h_incL);
                NewData LR_data = right_align_init_1d(n, d, h_incR);
                NewData Glc_r_data = left_align_init(m, k, d, h_incL);
                NewData Glc_i_data = left_align_init(m, k, d, h_incL);
                NewData Gcr_r_data = right_align_init(k, n, d, h_incR);
                NewData Gcr_i_data = right_align_init(k, n, d, h_incR);
                float* h_LL = LL_data.data;
                float* h_LR = LR_data.data;
                float* h_Glc_r = Glc_r_data.data;
                float* h_Glc_i = Glc_i_data.data;
                float* h_Gcr_r = Gcr_r_data.data;
                float* h_Gcr_i = Gcr_i_data.data;
                // Making sure remapping gives the same sizes as DataInit
                assert (sizeNewL == Glc_r_data.m);
                assert (sizeNewR == Gcr_r_data.n);
                // Moving lambdas and gammas to device
                float *d_LL, *d_LC, *d_LR, *d_Glc_r, *d_Gcr_r,*d_Glc_i, *d_Gcr_i;
                hipMalloc(&d_LL, sizeNewL * sizeof(float));
                hipMalloc(&d_LC, k * sizeof(float));
                hipMalloc(&d_LR, sizeNewR * sizeof(float));
                hipMalloc(&d_Glc_r, sizeNewL * k * sizeof(float));
                hipMalloc(&d_Glc_i, sizeNewL * k * sizeof(float));
                hipMalloc(&d_Gcr_r, k * sizeNewR * sizeof(float));
                hipMalloc(&d_Gcr_i, k * sizeNewR * sizeof(float));
                hipMemcpy(d_LL, h_LL, sizeNewL * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_LC, h_LC, k * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_LR, h_LR, sizeNewR * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_Glc_r, h_Glc_r, sizeNewL * k * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_Glc_i, h_Glc_i, sizeNewL * k * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_Gcr_r, h_Gcr_r, k * sizeNewR * sizeof(float), hipMemcpyDefault);
                hipMemcpy(d_Gcr_i, h_Gcr_i, k * sizeNewR * sizeof(float), hipMemcpyDefault);
                ////////////////////////////////////////
                // End of Lambda Gamma initialization //
                ////////////////////////////////////////

                // Initializing results array
                float *h_T_r, *h_T_i, *d_T_r, *d_T_i;
                hipHostMalloc(&h_T_r, sizeNewL * sizeNewR * sizeof(float));
                hipHostMalloc(&h_T_i, sizeNewL * sizeNewR * sizeof(float));
                hipMalloc(&d_T_r, sizeNewL * sizeNewR * sizeof(float));
                hipMalloc(&d_T_i, sizeNewL * sizeNewR * sizeof(float));

                hipEvent_t start, end;
                hipEventCreate(&start);
                hipEventCreate(&end);

                dim3 grid((sizeNewR + 63) / 64, (sizeNewL + 127) / 128);

                // warmup
                /*
                d: Maximum number of photons. Local (mode) dimensionality of the Hilbert space
                tau: the center charge.
                U: Unitary
                Glc: Gamma for left and center; Gcr: Gamma for center and right
                LL: Lambda for left; LC: lambda for center; LR: lambda for right
                cNewL: aligned charge for left; CC: sorted but unaligned charge for center; cNewR: aligned charge for right
                incC: charge increment index for center
                T: result theta matrix
                sizeNewL: m dimension after alignment; sizeNewR: n dimension after alignment
                */
                kernel<<<grid, 256>>>(
                    d, tau, d_U_r, d_U_i, d_Glc_r, d_Glc_i, d_Gcr_r, d_Gcr_i, d_LL, d_LC, d_LR, d_cNewL, d_CC, d_cNewR, d_incC, d_T_r, d_T_i, sizeNewL, sizeNewR, k, k * sizeof(float), sizeNewR * sizeof(float) * 8);

                hipEventRecord(start);
                for (int i = 0; i < n_iter; ++i) {
                    kernel<<<grid, 256>>>(
                        d, tau, d_U_r, d_U_i, d_Glc_r, d_Glc_i, d_Gcr_r, d_Gcr_i, d_LL, d_LC, d_LR, d_cNewL, d_CC, d_cNewR, d_incC, d_T_r, d_T_i, sizeNewL, sizeNewR, k, k * sizeof(float), sizeNewR * sizeof(float) * 8);
                }

                hipEventRecord(end);
                hipEventSynchronize(end);

                float ms;
                hipEventElapsedTime(&ms, start, end);
                hipEventDestroy(start);
                hipEventDestroy(end);

                long workload = n_iter * long(m) * n * k * 2;
                double gflops = (double(workload) / 1e9) / (double(ms) / 1e3);
                printf("Performance: %fGFLOPS; total time %fms\n", gflops, ms/n_iter);

                hipMemcpy(h_T_r, d_T_r, sizeNewL * sizeNewR * sizeof(float), hipMemcpyDefault);
                hipMemcpy(h_T_i, d_T_i, sizeNewL * sizeNewR * sizeof(float), hipMemcpyDefault);

                hipFree(d_CL);
                hipFree(d_CC);
                hipFree(d_CR);
                hipFree(d_idL);
                hipFree(d_idC);
                hipFree(d_idR);
                hipFree(d_incL);
                hipFree(d_incR);
                hipFree(d_incNewL);
                hipFree(d_incNewR);

                hipFree(d_U_r);
                hipFree(d_U_i);
                hipFree(d_Glc_r);
                hipFree(d_Glc_i);
                hipFree(d_Gcr_r);
                hipFree(d_Gcr_i);
                hipFree(d_LL);
                hipFree(d_LC);
                hipFree(d_LR);
                hipFree(d_cNewL);
                hipFree(d_CC);
                hipFree(d_cNewR);
                hipFree(d_incC);
                hipFree(d_T_r);
                hipFree(d_T_i);

                //chk = check(h_U, h_Glc, h_Gcr, h_LL, h_LC, h_LR, h_T, sizeNewL, sizeNewR, k);
                chk = check(h_T_r, h_T_i, d, tau, h_U_r, h_U_i, h_Glc_r, h_Glc_i, h_Gcr_r, h_Gcr_i, h_LL, h_LC, h_LR, h_cNewL, h_CC, h_cNewR, sizeNewL, sizeNewR, k); 
                if (!chk) { printf("Failed at m %i, n %i, k %i", m, n, k); }
                //save results to file
                if (chk) {
                    save((std::string)"./out/U_r.npy", h_U_r, d, d, d);
                    save((std::string)"./out/U_i.npy", h_U_i, d, d, d);
                    save((std::string)"./out/Glc_r.npy", h_Glc_r, sizeNewL, k);
                    save((std::string)"./out/Glc_i.npy", h_Glc_i, sizeNewL, k);
                    save((std::string)"./out/Gcr_r.npy", h_Gcr_r, k, sizeNewR);
                    save((std::string)"./out/Gcr_i.npy", h_Gcr_i, k, sizeNewR);
                    save((std::string)"./out/LL.npy", h_LL, sizeNewL);
                    save((std::string)"./out/LC.npy", h_LC, k);
                    save((std::string)"./out/LR.npy", h_LR, sizeNewR);
                    save((std::string)"./out/CL.npy", h_cNewL, sizeNewL);
                    save((std::string)"./out/CC.npy", h_CC, k);
                    save((std::string)"./out/CR.npy", h_cNewR, sizeNewR);
                    save((std::string)"./out/incC.npy", h_incC, d);
                    save((std::string)"./out/T_r.npy", h_T_r, sizeNewL, sizeNewR);
                    save((std::string)"./out/T_i.npy", h_T_i, sizeNewL, sizeNewR);
                }

                hipHostFree(h_CL);
                hipHostFree(h_CC);
                hipHostFree(h_CR);
                hipHostFree(h_idL);
                hipHostFree(h_idR);
                hipHostFree(h_incL);
                hipHostFree(h_incC);
                hipHostFree(h_incR);
                hipHostFree(h_incNewL);
                hipHostFree(h_incNewR);

                hipHostFree(h_U_r);
                hipHostFree(h_U_i);
                hipHostFree(h_Glc_r);
                hipHostFree(h_Glc_i);
                hipHostFree(h_Gcr_r);
                hipHostFree(h_Gcr_i);
                hipHostFree(h_LL);
                hipHostFree(h_LC);
                hipHostFree(h_LR);
                hipHostFree(h_cNewL);
                hipHostFree(h_CC);
                hipHostFree(h_cNewR);
                hipHostFree(h_T_r);
                hipHostFree(h_T_i);
            }
        }
    }
}